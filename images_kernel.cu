#include "hip/hip_runtime.h"
extern "C" {
    #include "images.h"
}

__global__ void resizeGPU(const unsigned char*src,int srcWidth,int srcHeight,
                          unsigned char *dst_resize,int dstWidth,int dstHeight,
                          float w_ratio,float h_ratio,float *dst)
{
	const int x = blockIdx.x*blockDim.x+threadIdx.x;
	const int y = blockIdx.y*blockDim.y+threadIdx.y;
	if(x < dstWidth && y < dstHeight){
        float srcXf=  x * w_ratio;
        float srcYf =  y * h_ratio;
        int srcX = (int)srcXf;
        int srcY = (int)srcYf;
        float u= srcXf - srcX;
        float v = srcYf - srcY;
        int dstOffset = (y*dstWidth + x);
        int offset;
        //resize(inter_linear)
        for(int i = 0;i < 3;++i){
            offset = dstOffset*3 + i;
            dst_resize[offset] = (1-u)*(1-v)*src[(srcY*srcWidth+srcX)*3 + i];
            dst_resize[offset] += (1-u)*v*src[((srcY+1)*srcWidth+srcX)*3 + i];
            dst_resize[offset] += u*(1-v)*src[(srcY*srcWidth+srcX+1)*3 + i];
            dst_resize[offset] += u*v*src[((srcY+1)*srcWidth+srcX+1)*3 + i];
        }

        // BGR_RGB and channel split.
        int image_patch = dstWidth*dstHeight;
        offset = dstOffset;
//        BGR2RGB
//        dst[offset*3 + 2] = dst_resize[offset*3];
//        dst[offset*3 + 1] = dst_resize[offset*3+1];
//        dst[offset*3] = dst_resize[offset*3+2];
        //merge

        dst[offset] = dst_resize[offset*3+2] / 255.0f;
        dst[offset + image_patch] = dst_resize[offset*3+1] / 255.0f;
        dst[offset + image_patch*2] = dst_resize[offset*3]  / 255.0f;

	}
}

Image* resize_cu(const unsigned char*src,Image* img){

    hipMemcpy(img->cu_src,src,img->srcWidth*img->srcHeight*3*sizeof(unsigned char),hipMemcpyHostToDevice);
    int uint = 16;
    dim3 grid((img->dstWidth+uint-1)/uint,(img->dstHeight+uint-1)/uint);
    dim3 block(uint,uint);
    float w_ratio = (float)img->srcWidth/img->dstWidth;
    float h_ratio = (float)img->srcHeight/img->dstHeight;

    resizeGPU<<<grid,block>>>(img->cu_src, img->srcWidth,img->srcHeight,
    img->cu_dst_resize,img->dstWidth,img->dstHeight,w_ratio,h_ratio,img->cu_dst);
    hipMemcpy(img->data,img->cu_dst,img->dstHeight*img->dstWidth*3*sizeof(float),hipMemcpyDeviceToHost);
    return img;

}















